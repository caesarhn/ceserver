#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void neuralNetwork(int *a, int *b, int *c, int n){
    int index = threadIdx.x; // Mendapatkan indeks thread
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

extern "C" void runKernel(){
    int n = 512;
    int size = n * sizeof(int);

    // Alokasi memori di CPU (host)
    int *a = (int *)malloc(size);
    int *b = (int *)malloc(size);
    int *c = (int *)malloc(size);

    // Inisialisasi data
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Alokasi memori di GPU (device)
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Salin data dari CPU ke GPU
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Jalankan kernel dengan jumlah thread sesuai ukuran array
    neuralNetwork<<<1, n>>>(d_a, d_b, d_c, n);

    // Salin hasil dari GPU kembali ke CPU
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Tampilkan hasil
    printf("Hasil penjumlahan:\n");
    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Bebaskan memori
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}